#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <algorithm>
#include <memory>
#include <fstream>
#include <string>

#define N 1024 // size for N x N matrices

__global__ void MatrixMultiplication(float* M, float* N, float* P, int Width) 
{
	int size = Width * Width * sizeof(float);
	float* Md, Nd, Pd;
	
	// 2D Thread ID
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// product stores the Pd element that is computed by the thread
	float product = 0;

	// Allocate device memory for M, N, and P
	hipMalloc((void**)Md, size);
	hipMalloc((void**)Nd, size);
	hipMalloc((void**)Pd, size);

	// copy M and N to allocated device memory locations
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	hipMemcpy(Pd, P, size, hipMemcpyHostToDevice);

	// Kernel invocation code - to have the device to perform the actual matrix multiplication
	for (int i = 0; i < Width; ++i)
	{
		float MdElement = Md[ty * Width + i]; // incrementing horizontally on a 2d rep.
		float NdElement = Nd[i * Width + tx]; // incrementing downwards on a 2d rep.
		product += MdElement * NdElement;
	}

	// Write the matrix to device memory each thread writes one element
	Pd[ty * Width + tx] = product;

	// copy P from the device memory
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	
	// Free device matrices
	hipFree(Md); hipFree(Nd); hipFree(Pd); 
}

float* initialize(float* arr, int size) 
{
	for (int i = 0; i < size; i++) 
	{
		arr[i] = 1.0f;
	}
	return arr;
}

void validate()
{
}

int main(void) 
{
	int matrixSize = N * N;

	// Allocate and initialize the matrices M, N, P
	float Mh[N * N * sizeof(float);
	float Nh[N * N * sizeof(float);
	float Ph[N * N * sizeof(float);

	Mh = initialize(Mh, matrixSize);
	Nh = initialize(Nh, matrixSize);
	Ph = initialize(Ph, matrixSize);

	// I/O to read the input matrices M and N

	// M * N on the device
	
	// I/O to write the output matrix P
	// Free matrices M, N, P

	return 0;
}
