#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <vector>

using std::accumulate;
using std::cout;
using std::generate;
using std::vector;

#define SHMEM_SIZE 256

// Reduction 0: Interleaved Addressing
__global__ void reduce0(int *g_idata, int *g_odata) {

    extern __shared__ int sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared mem
    /*
        Problem: highly divergent warps are very inificient and % operation is expensive
            - Warp divergence occurs when two threads of the same warp diverge in their execution due 
                to a branch instruction, where one thread branches and the other does not. 
                This leads to serialization of the two threads by the CUDA hardware until their execution
                path converges again.
    */
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

// Reduction 1: Interleaved Addressing
__global__ void reduce0(int *g_idata, int *g_odata) {

    extern __shared__ int sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared mem
    /*
        Solution(reduction 0): use strided index to avoid divergence
        New Problem: shared memory bank conflicts
            - A bank conflict appears when two or more threads belonging to the same warp 
            simultaneously request access to different shared memory locations in the same 
            bank. Conflicting memory accesses cannot be handled in the same memory transaction.
    */
    for (unsigned int s=1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

// Reduction 2: Sequential Addressing
__global__ void reduce2(int *g_idata, int *g_odata) {

    extern __shared__ int sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared mem
    /*
        Solution(reduction 1): use reversed loop and threadId based indexing 
        New Problem: Half of the threads are idle during the first iteration 
    */
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

// Reduction 3: First Add During Load
__global__ void reduce3(int *g_idata, int *g_odata) {

    extern __shared__ int sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();

    // do reduction in shared mem
    /*
        Solution(reduction 1): use reversed loop and threadId based indexing 
        New Problem: Half of the threads are idle during the first iteration 
    */
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

int main() {
    // Array size of 2^16 (65536 elements)
    constexpr int N = 1 << 16;
    constexpr size_t bytes = sizeof(int) * N;

    // Vectors for holding the host-side (CPU-side) data
    std::vector<int> a;
    a.reserve(N);
    std::vector<int> b;
    b.reserve(N);
    std::vector<int> c;
    c.reserve(N);

    // Initialize random numbers in each array
    for (int i = 0; i < N; i++) {
        a.push_back(rand() % 100);
        b.push_back(rand() % 100);
    }

    // Allocate memory on the device
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy data from the host to the device (CPU -> GPU)
    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

    // Threads per CTA (1024)
    int NUM_THREADS = 1 << 10;

    // CTAs per Grid
    // We need to launch at LEAST as many threads as we have elements
    // This equation pads an extra CTA to the grid if N cannot evenly be divided
    // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

    // Launch the kernel on the GPU
    // Kernel calls are asynchronous (the CPU program continues execution after
    // call, but no necessarily before the kernel finishes)
    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

    // Copy sum vector from device to host
    // hipMemcpy is a synchronous operation, and waits for the prior kernel
    // launch to complete (both go to the default stream in this case).
    // Therefore, this hipMemcpy acts as both a memcpy and synchronization
    // barrier.
    hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    // Check result for errors
    verify_result(a, b, c);

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "COMPLETED SUCCESSFULLY\n";

    return 0;
    }